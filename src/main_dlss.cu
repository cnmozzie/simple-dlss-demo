#include "hip/hip_runtime.h"
#ifdef _WIN32
  #include <GL/gl3w.h>
#else
  #include <GL/glew.h>
#endif
#include <GLFW/glfw3.h>
#include <tiny-cuda-nn/multi_stream.h>
#include <tiny-cuda-nn/common.h>
#include <Eigen/Dense>

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/dlss.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/npy.hpp>

void simple_glfw_error_callback(int error, const char* description) 
{
    std::cout << "GLFW error #" << error << ": " << description << std::endl;
}

__global__ void dlss_prep_kernel(
	Eigen::Vector2i resolution,
	float* depth_buffer,
	hipSurfaceObject_t depth_surface,
	hipSurfaceObject_t mvec_surface,
	hipSurfaceObject_t exposure_surface
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	uint32_t x_orig = x;
	uint32_t y_orig = y;

	const float depth = depth_buffer[idx];
	Eigen::Vector2f mvec = {0., 0.}; // motion vector

	surf2Dwrite(make_float2(mvec.x(), mvec.y()), mvec_surface, x_orig * sizeof(float2), y_orig);

	// Scale depth buffer to be guaranteed in [0,1].
	surf2Dwrite(std::min(std::max(depth / 128.0f, 0.0f), 1.0f), depth_surface, x_orig * sizeof(float), y_orig);

	// First thread write an exposure factor of 1. Since DLSS will run on tonemapped data,
	// exposure is assumed to already have been applied to DLSS' inputs.
	if (x_orig == 0 && y_orig == 0) {
		surf2Dwrite(1.0f, exposure_surface, 0, 0);
	}
}

void render_frame(ngp::CudaRenderBuffer& render_buffer) 
{
    std::cout << "render frame begin" << std::endl;
    
    // CUDA stuff
	tcnn::StreamAndEvent m_stream;
    render_buffer.clear_frame(m_stream.get());
    render_buffer.set_color_space(ngp::EColorSpace::Linear);
	render_buffer.set_tonemap_curve(ngp::ETonemapCurve::Identity);

    const std::string path{"depth.npy"};
    std::cout << "load depth buffer..." << std::endl;
	std::vector<float> data;
	std::vector<unsigned long> shape;
	bool is_fortran;
	npy::LoadArrayFromNumpy(path, shape, is_fortran, data);
    std::cout << "buffer size: " << data.size() << std::endl;
	render_buffer.host_to_depth_buffer(data);

    // Prepare DLSS data: motion vectors, scaled depth, exposure
    std::cout << "prepare the dlss data..." << std::endl;
    auto res = render_buffer.in_resolution();
    //bool distortion = false;
    const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { tcnn::div_round_up((uint32_t)res.x(), threads.x), tcnn::div_round_up((uint32_t)res.y(), threads.y), 1 };
    float m_dlss_sharpening = 0.0;
    dlss_prep_kernel<<<blocks, threads, 0, m_stream.get()>>>(
			res,
			render_buffer.depth_buffer(),
			render_buffer.dlss()->depth(),
			render_buffer.dlss()->mvec(),
			render_buffer.dlss()->exposure()
	);
    render_buffer.set_dlss_sharpening(m_dlss_sharpening);

    std::cout << "run dlss..." << std::endl;
    float m_exposure = 0.0;
    Eigen::Array4f m_background_color = {0.0f, 0.0f, 0.0f, 1.0f};
    render_buffer.accumulate(m_exposure, m_stream.get());
    render_buffer.tonemap(m_exposure, m_background_color, ngp::EColorSpace::Linear, m_stream.get());
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
}

int main() 
{
    std::cout << "custom glfw init" << std::endl;
    glfwSetErrorCallback(simple_glfw_error_callback);
    if (!glfwInit()) {
		throw std::runtime_error{"GLFW could not be initialized."};
	}
    std::cout << "custom enable dlss" << std::endl;
    try {
		ngp::vulkan_and_ngx_init();
	} catch (const std::runtime_error& e) {
		tlog::warning() << "Could not initialize Vulkan and NGX. DLSS not supported. (" << e.what() << ")";
	}

    int in_height = 640;
	int in_width = 380;
    ngp::CudaRenderBuffer m_windowless_render_surface{std::make_shared<ngp::CudaSurface2D>()};
    m_windowless_render_surface.resize({in_width, in_height});
	m_windowless_render_surface.reset_accumulation();

    unsigned long out_height = 1080;
	unsigned long out_width = 1920;
    // enable dlss
	tlog::info() << "custom enable dlss for render buffer";
	m_windowless_render_surface.enable_dlss({out_width, out_height});
	auto render_res = m_windowless_render_surface.in_resolution();
	if (m_windowless_render_surface.dlss()) {
		render_res = m_windowless_render_surface.dlss()->clamp_resolution(render_res);
	}
	m_windowless_render_surface.resize(render_res);

    render_frame(m_windowless_render_surface);

    std::cout << "begin to transfer data..." << std::endl;

    //float *result = (float*)malloc(sizeof(float)*out_height*out_width*4);
    std::vector<float> result(out_height*out_width*4, 0.0);

    hipError_t x = hipMemcpy2DFromArray(&result[0], out_width * sizeof(float) * 4, m_windowless_render_surface.surface_provider().array(), 0, 0, out_width * sizeof(float) * 4, out_height, hipMemcpyDeviceToHost);
    CUDA_CHECK_THROW(x);

    const std::vector<long unsigned> shape{out_height, out_width, 4};
	const bool fortran_order{false};
    const std::string path{"out.npy"};
	
	// try to save frame_buffer here?
	std::cout << "save frame buffer..." << std::endl;
	npy::SaveArrayAsNumpy(path, fortran_order, shape.size(), shape.data(), result);

    //CUDA_CHECK_THROW(hipMemcpy2DToArray(m_windowless_render_surface.surface_provider().array(), 0, 0, result, out_width * sizeof(float) * 4, out_width * sizeof(float) * 4, out_height, hipMemcpyDeviceToHost));
    //free(result);

    return 0;
}